// Simple Base and Size Table (BST) implementation for memory tracking in CUDA applications
#include <iostream>
#include <unordered_map>
#include <random>
#include <mutex>

// Metadata structure for each allocation
struct Metadata {
    void *base_addr;
    size_t size;
    uint8_t tag; // 4-bit tag for temporal safety
};

class BSTracker {
private:
    std::unordered_map<void*, Metadata> bst;
    std::mutex bst_mutex;
    std::mt19937 rng;
    std::uniform_int_distribution<uint8_t> tag_dist;

public:
    BSTracker() : rng(std::random_device{}()), tag_dist(1, 15) {}

    // Allocate memory and store metadata
    void* allocate(size_t size) {
        void* ptr = nullptr;
        hipMalloc(&ptr, size);
        if (ptr) {
            Metadata meta = {ptr, size, tag_dist(rng)};
            std::lock_guard<std::mutex> lock(bst_mutex);
            bst[ptr] = meta;
            std::cout << "Allocated " << size << " bytes at " << ptr << " with tag " << (int)meta.tag << "\n";
        }n
        return ptr;
    }


    // Free memory and invalidate metadata
    void deallocate(void* ptr) {
        std::lock_guard<std::mutex> lock(bst_mutex);
        auto it = bst.find(ptr);
        if( it != bst.end() ) {
            it->second.tag = 0; // Invalidate tag
            hipFree(ptr);
            bst.erase(it);
            std::cout << "Deallocate at " << ptr << "\n";
        } else {
            std::cout << "Attempted to free untracked pointer " << ptr << "\n";
        }
    }

    // Check if access is within bounds
    bool check_bounds(void* ptr, size_t access_size) {
        std::lock_guard<std::mutex> lock(bst_mutex);
        for (const auto& [base, meta] : bst) {
            if (ptr >= base && (char*)ptr + access_size <= (char*)base + meta.size) {
                return true; // Valid access
            }
        }
        return false; // Out-of-bounds access
    }

};


// Example usage
int main() {
    BSTracker tracker;
    void* ptr = tracker.allocate(1024);

    if (tracker.check_bounds(ptr, 512)) {
        std::cout << "Access within bounds." << std::endl;
    } else {
        std::cerr << "Out-of-bounds access detected!" << std::endl;
    }

    tracker.deallocate(ptr);
    return 0;
}
